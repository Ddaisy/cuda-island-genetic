#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <hip/hip_runtime.h>

const int CHUNKS = 4;
const int GENERATIONS = 10;


const int CHECK_VALUES_EVERY = 5;
const int SHOW_ALL_VALUES = 0;


const int ISLANDS_PER_ROW = 16;
const int GENOME_LENGTH=10;
const int BLOCKS_PER_ROW = 16;
const int ISLAND_POPULATION=100;
const int SELECTION_COUNT=80;
const float MUTATION_CHANCE= 0.8;
const int ITEMS_MAX_WEIGHT = 5;
const int ITEMS_MAX_VALUE = 20;
const int ITEMS_MAX = 20;



__device__ float
fitnessValue(float *baseSizes, float *baseValues, unsigned char *phenotype, float backpackMaxSize)
{
    float size = 0, value = 0;
    int count = 0;

    for( int i = 0; i < GENOME_LENGTH; i++ ) {
    	count = phenotype[i];
    	size += baseSizes[i] * count;
    	value += baseValues[i] * count;
    }

    if(size > backpackMaxSize) {
    	return 0.0;
    }
    return value;
}

__device__ void
sortByFitness(float*populationFitness, unsigned char* sortedAssoc, float* totalFitness)
{
    int i, j;
    *totalFitness = 1;
    float phenotypeFitness = 0;
    for ( i = 0; i < ISLAND_POPULATION; ++i ){
    	sortedAssoc[i] = i;
        phenotypeFitness = populationFitness[i];
        for (
        		j = i;
        		j > 0 && populationFitness[sortedAssoc[j - 1]] > phenotypeFitness;
        		j-- )
        {
        	sortedAssoc[j] = sortedAssoc[j - 1];
        }
        sortedAssoc[j] = i;
    }
}

__device__ void
normalizeFitness(float*populationFitness, unsigned char* sortedAssoc, float totalFitness)
{
    int i, j;
    float lastFitness = 0;
    for ( i = 0; i < ISLAND_POPULATION; ++i ){
    	j = sortedAssoc[i];
        lastFitness += populationFitness[j];
        populationFitness[j] = lastFitness/totalFitness;
    }
}

__device__ void
selectionTrunc(unsigned char* sortedAssoc, unsigned char* selectedAssoc)
{
	for(int i = 1; i <= SELECTION_COUNT; i++) {
		selectedAssoc[i-1] = sortedAssoc[ISLAND_POPULATION - i];
	}
}

__device__ float
computeFitnessValue(float *baseSizes, float *baseValues, unsigned char *populationRow, float*populationFitness, float backpackMaxSize)
{
    float max = 0;
    for(int i = 0; i < ISLAND_POPULATION; i++ ) {
    	populationFitness[i] = fitnessValue(
    			baseSizes,
    			baseValues,
    			&(populationRow[GENOME_LENGTH * i]),
    			backpackMaxSize);
    	if( populationFitness[i] > max) {
    		max = populationFitness[i];
    	}
    }
    return max;
}

__device__ void
crossover(
		unsigned char *populationRow,
		unsigned char *newPopulation,
		unsigned char *selectedPopulation,
		hiprandState_t *randomState)
{
	int i,j;
	int selectedPhenotype,
		selectedPhenotypeA,
		selectedPhenotypeB;
	int treshold = 0;
	for( i = 0; i < ISLAND_POPULATION; i++) {

		selectedPhenotypeA = selectedPopulation[ hiprand(randomState) % SELECTION_COUNT ];
		selectedPhenotypeB = selectedPopulation[ hiprand(randomState) % SELECTION_COUNT ];

		treshold = hiprand(randomState) % GENOME_LENGTH;

		for(j = 0; j < GENOME_LENGTH; j++) {
			if(j < treshold) {
				selectedPhenotype = selectedPhenotypeA;
			} else {
				selectedPhenotype = selectedPhenotypeB;
			}

			newPopulation[i * GENOME_LENGTH + j] =
					populationRow[selectedPhenotype * GENOME_LENGTH];
		}
	}
}

__device__ void
mutation(
		unsigned char *newPopulation,
		hiprandState_t *randomState)
{
	int i;

	for( i = 0; i < ISLAND_POPULATION; i++) {
		if(hiprand_uniform(randomState) < MUTATION_CHANCE) {
			newPopulation[ i* GENOME_LENGTH + (hiprand(randomState) % GENOME_LENGTH ) ]
			               = hiprand(randomState) % ITEMS_MAX;
		}
	}
}

__device__ void
killPreviousPopulation(
		unsigned char *populationRow,
		unsigned char *newPopulation
)
{
	int i;

	for( i = 0; i < ISLAND_POPULATION * GENOME_LENGTH; i++) {
		populationRow[i] = newPopulation[i];
	}
}


__global__ void
geneticAlgorithmGeneration(
	hiprandState_t* states, 
	float *baseSizes, 
	float *baseValues, 
	unsigned char *population,
	float* bestValues, 
	float backpackMax
) 
{
	float backpackMaxSize = backpackMax;

	//index of the island itself
    int island_y = blockDim.y * blockIdx.y + threadIdx.y;
    int island_x = blockDim.x * blockIdx.x + threadIdx.x;

    unsigned char * populationRow = &population[island_y * GENOME_LENGTH * ISLAND_POPULATION * ISLANDS_PER_ROW + island_x * GENOME_LENGTH * ISLAND_POPULATION ];

    __shared__ float sharedbaseSizes[GENOME_LENGTH];
    __shared__ float sharedbaseValues[GENOME_LENGTH];

	__shared__ hiprandState_t randomState;

	randomState = states[blockDim.x*blockDim.y];
	int i = threadIdx.x*threadIdx.y;
    if(i < GENOME_LENGTH){
    	sharedbaseSizes[i] = baseSizes[i];
    	sharedbaseValues[i] = baseValues[i];
    }
    __syncthreads();

	float populationFitness[ISLAND_POPULATION];

	float best = computeFitnessValue(
			sharedbaseSizes,
			sharedbaseValues,
			populationRow,
			populationFitness,
			backpackMaxSize);

	bestValues[island_y * ISLANDS_PER_ROW + island_x] = best;

	unsigned char sortAssoc[ISLAND_POPULATION];
	float totalFitness;

	sortByFitness(populationFitness, sortAssoc, &totalFitness);
	//normalizeFitness(populationFitness, sortAssoc, totalFitness);

	unsigned char selectedAssoc[SELECTION_COUNT];
	selectionTrunc(sortAssoc, selectedAssoc);

	unsigned char  newPopulation[ISLAND_POPULATION*GENOME_LENGTH];
	crossover(populationRow, newPopulation, selectedAssoc, &randomState);
	mutation(newPopulation, &randomState);
	killPreviousPopulation(populationRow, newPopulation);
}

/* this GPU kernel function is used to initialize the random states */
__global__ void init(unsigned int seed, hiprandState_t* states) {

  /* we have to initialize the state */
  hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
		  blockDim.y * blockIdx.y , /* the sequence number should be different for each core (unless you want all
                             cores to get the same sequence of numbers for some reason - use thread id! */
              0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
              &states[blockDim.y * blockIdx.y ]);
}

/* this GPU kernel function is used to initialize the random states */
__global__ void randomizePopulation(hiprandState_t* states, unsigned char* population ) {
	int island_y = blockDim.y * blockIdx.y + threadIdx.y;
	int island_x = blockDim.x * blockIdx.x + threadIdx.x;

	__shared__ hiprandState_t randomState;
	randomState = states[blockDim.y * blockIdx.y ];

	unsigned char * populationRow = &population[island_y * GENOME_LENGTH * ISLAND_POPULATION * ISLANDS_PER_ROW + island_x * GENOME_LENGTH * ISLAND_POPULATION ];

	for(int i = 0; i < GENOME_LENGTH * ISLAND_POPULATION; i++) {
		populationRow[i] = hiprand(&randomState) % ITEMS_MAX;
	};
}

/**
 * Host main routine
 */
int
main(void)
{
    hipError_t err = hipSuccess;
    int ISLANDS = ISLANDS_PER_ROW * ISLANDS_PER_ROW * BLOCKS_PER_ROW * BLOCKS_PER_ROW;

    srand(time(NULL));

    int sizeFloat = sizeof(float);
    int sizeInt = sizeof(unsigned char);

    int baseLength =  GENOME_LENGTH;
    int sizeBase =  baseLength * sizeFloat;
    int populationLength = ISLANDS * GENOME_LENGTH * ISLAND_POPULATION;
    int sizePopulation = populationLength * sizeInt;
    int sizeBestValue = ISLANDS * sizeFloat;

    int blocksPerGrid = BLOCKS_PER_ROW*BLOCKS_PER_ROW;


    float *cu_baseSizes = NULL;
    err = hipMalloc((void **)&cu_baseSizes, sizeBase);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector baseSize (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    float *cu_baseValues = NULL;
    err = hipMalloc((void **)&cu_baseValues, sizeBase);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector baseValues (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    unsigned char *cu_populationA = NULL;
    err = hipMalloc((void **)&cu_populationA, sizePopulation);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector Population (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    unsigned char *cu_populationB = NULL;
    err = hipMalloc((void **)&cu_populationB, sizePopulation);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector Population (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    float *cu_bestValue = NULL;
    err = hipMalloc((void **)&cu_bestValue, sizeBestValue);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector bestValue (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    hiprandState_t *states = NULL;
    /* allocate space on the GPU for the random states */
    err = hipMalloc((void**) &states, blocksPerGrid * sizeof(hiprandState_t));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector randomStates (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    float *baseSizes = (float*)malloc(sizeBase);
    float *baseValues = (float*)malloc(sizeBase);
    unsigned char *population = (unsigned char *)malloc(sizePopulation);
    float *bestValue = (float*)malloc(sizeBestValue);





    if (baseSizes == NULL || baseValues == NULL || population == NULL || bestValue == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    for(int i = 0; i <baseLength; i++) {
    	baseSizes[i] = (float)rand()/(float)(RAND_MAX/ITEMS_MAX_WEIGHT);
    	baseValues[i] =  (float)rand()/(float)(RAND_MAX/ITEMS_MAX_VALUE);
    	printf("%f - %f |", baseSizes[i], baseValues[i] );
    }
	printf("\n");

	/*
    for(int i = 0; i <populationLength; i++) {
    	population[i] = rand() % ITEMS_MAX;
    	printf("%d - ", population[i]);
    }
    */

    err = hipMemcpy( cu_baseSizes, baseSizes, sizeBase, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector baseSizes from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy( cu_baseValues, baseValues, sizeBase, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector baseValues from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    /*
    err = cudaMemcpy( cu_population, population, sizePopulation, cudaMemcpyHostToDevice);
    if (err != cudaSuccess)
    {
        fprintf(stderr, "Failed to copy vector population from host to device (error code %s)!\n", cudaGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    */


    dim3 dimGrid;
    dimGrid.x = BLOCKS_PER_ROW;
    dimGrid.y = BLOCKS_PER_ROW;

    dim3 dimBlock;
    dimBlock.x = ISLANDS_PER_ROW;
    dimBlock.y = ISLANDS_PER_ROW;
    /* invoke the GPU to initialize all of the random states */

    printf("CUDA Init kernel launch with %d blocks of %d threads\n", blocksPerGrid, dimBlock.x * dimBlock.y);
    init<<<dimGrid, 1>>>(time(0), states);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch Init kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    int backpackMaxSize = ITEMS_MAX*2;

    hipStream_t stream1, stream2;
    hipStreamCreate ( &stream1) ;
    hipStreamCreate ( &stream2) ;
    int chunkTargetId = 0;
	char chunkFileName[20];
	char chunkTargetFileName[20];

    unsigned char *cu_populationLoad = NULL;
    unsigned char *cu_populationUse = NULL;

    bool useA = false;
	float maxTotal = 0;

    printf("Genetic algorithm launch with %d blocks of %d threads\n", dimGrid.x*dimGrid.y, dimBlock.x * dimBlock.y);
    for( int i = 1; i <= GENERATIONS ; i++) {

    	for(int k = 0; k < CHUNKS; k++) {
    		useA = !useA;
    		if( useA ) {
    		    cu_populationUse = cu_populationA;
    		    cu_populationLoad = cu_populationB;
    		} else {
    		    cu_populationUse = cu_populationB;
    		    cu_populationLoad = cu_populationA;
    		}
    		sprintf(chunkFileName, "chunk%d.data", k);
    		if(i == 1) {
    		    randomizePopulation<<<dimGrid, dimBlock, 0, stream1>>>( states, cu_populationUse);
    			err = hipGetLastError();
    			if (err != hipSuccess)
    			{
    				fprintf(stderr, "Failed to launch randomizePopulation kernel (error code %s)!\n", hipGetErrorString(err));
    				exit(EXIT_FAILURE);
    			}
    		}
    		if(i !=1 || k+1 == CHUNKS){

    			//load data for the next chunk
    			chunkTargetId = (k+1) % CHUNKS;
        		sprintf(chunkTargetFileName, "chunk%d.data", chunkTargetId);
    			FILE *ifp = fopen(chunkTargetFileName, "rb");
    			fread(population, sizeof(char), sizePopulation, ifp);
        		err = hipMemcpyAsync(cu_populationLoad, population, sizePopulation, hipMemcpyHostToDevice, stream2);
    			if (err != hipSuccess)
    			{
    				fprintf(stderr, "Failed to copy data TO device (error code %s)!\n", hipGetErrorString(err));
    				exit(EXIT_FAILURE);
    			}
    		}
			geneticAlgorithmGeneration<<<dimGrid, dimBlock, 0, stream1>>>(
					states,
					cu_baseSizes,
					cu_baseValues,
					cu_populationUse,
					cu_bestValue,
					backpackMaxSize);
			hipDeviceSynchronize();
			err = hipGetLastError();
			if (err != hipSuccess)
			{
				fprintf(stderr, "Failed to launch geneticAlgorithmGeneration kernel (error code %s)!\n", hipGetErrorString(err));
				exit(EXIT_FAILURE);
			}


    		err = hipMemcpy(population, cu_populationUse, sizePopulation, hipMemcpyDeviceToHost);
			if (err != hipSuccess)
			{
				fprintf(stderr, "Failed to copy data FROM device (error code %s)!\n", hipGetErrorString(err));
				exit(EXIT_FAILURE);
			}
    		FILE *f = fopen(chunkFileName, "wb");
    		fwrite(population, sizeof(char), sizePopulation, f);
    		fclose(f);

			if( i % CHECK_VALUES_EVERY == 0 ) {
				float max = 0;
				// Verify that the result vector is correct
				err = hipMemcpy(bestValue, cu_bestValue, sizeBestValue, hipMemcpyDeviceToHost);
				if (err != hipSuccess)
				{
					fprintf(stderr, "Failed to copy best values from device (error code %s)!\n", hipGetErrorString(err));
					exit(EXIT_FAILURE);
				}
				for (int i = 0; i < ISLANDS; ++i)
				{
					if(SHOW_ALL_VALUES == 1) {
						printf("%f | ", bestValue[i]);
					}
					if(bestValue[i] > max) {
						max = bestValue[i];
					}
					if(bestValue[i] > maxTotal) {
						maxTotal = bestValue[i];
					}

				}
				printf("\nMax %d: %f\n",k, max);
				printf("\n");
			}
    	}

		if( i % CHECK_VALUES_EVERY == 0 ) {
			printf("\nMaxTotal %d: %f\n",i, maxTotal);
			printf("\n");
		}
    }

    // Free device global memory
    err = hipFree(cu_baseSizes);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Free device global memory
    err = hipFree(cu_baseValues);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Free device global memory
    err = hipFree(cu_populationA);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Free device global memory
    err = hipFree(cu_populationB);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Free device global memory
    err = hipFree(cu_bestValue);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Free device global memory
    err = hipFree(states);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(baseSizes);
    free(baseValues);
    free(population);
    free(bestValue);

    // Reset the device and exit
    // cudaDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling cudaDeviceReset causes all profile data to be
    // flushed before the application exits
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    for(int x = 0; x < CHUNKS; x++) {
    	sprintf(chunkFileName, "chunk%d.data", x);
    	remove(chunkFileName);
    }

    printf("Done\n");
    return 0;
}

